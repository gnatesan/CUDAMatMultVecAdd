#include "hip/hip_runtime.h"
#include <iostream> 
#include <math.h>
#include <time.h>

__global__ void add(int n, float *x, float *y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;
	for (int i = index; i < n; i += stride) {
		y[i] = x[i] + y[i];
	}
}

int main(int args, char* argv[]) {
	int K = atoi(argv[1]);
	int numElements = K * 1000000;
	size_t size = K * 1000000 * sizeof(float);
	float *x, *y;
	
	float *hx = (float*)malloc(size);
	float *hy = (float*)malloc(size);

	// initialize x and y arrays on the host
	for (int i = 0; i < K; i++) {
		hx[i] = 1.0f;
		hy[i] = 2.0f;
	}	

	//Allocate vectors in device memory
	hipMalloc(&x, size);
	hipMalloc(&y, size);


 	// Copy vectors from host memory to device global memory
 	hipMemcpy(x, hx, size, hipMemcpyHostToDevice);
 	hipMemcpy(y, hy, size, hipMemcpyHostToDevice);


	float elapsed_time = 0.0;
	struct timespec start, end;

	//1 BLOCK 1 THREADS 

	int blockSize = 1;
	int numBlocks = 1;

	// Invoke kernel for warm up
	add<<<numBlocks, blockSize>>>(numElements, x, y);


	// Synchronize to make sure everyone is done in the warmup.
  	hipDeviceSynchronize();

  	// Set up timer
  	clock_gettime(CLOCK_MONOTONIC, &start);

	add<<<numBlocks, blockSize>>> (numElements, x, y);

	// Synchronize to make sure everyone is done.
  	hipDeviceSynchronize();	

	// Compute and report the timing results
  	clock_gettime(CLOCK_MONOTONIC, &end);
	elapsed_time = ((float)end.tv_sec - (float)start.tv_sec) + ((float)end.tv_nsec - (float)start.tv_nsec) / 1000000000.0;

	printf("K=%d Number of elements in array=%d Number of blocks=%d\nBlock size=%d  Execution time=%lf seconds\n\n", K, numElements, numBlocks, blockSize, elapsed_time);






	//1 BLOCK 256 THREADS 

	blockSize = 256;
	numBlocks = 1;

	// Synchronize to make sure everyone is done in the warmup.
  	hipDeviceSynchronize();

  	// Set up timer
  	clock_gettime(CLOCK_MONOTONIC, &start);

	add<<<numBlocks, blockSize>>> (numElements, x, y);

	// Synchronize to make sure everyone is done.
  	hipDeviceSynchronize();	

	// Compute and report the timing results
  	clock_gettime(CLOCK_MONOTONIC, &end);
	elapsed_time = ((float)end.tv_sec - (float)start.tv_sec) + ((float)end.tv_nsec - (float)start.tv_nsec) / 1000000000.0;

	printf("K=%d Number of elements in array=%d Number of blocks=%d\nBlock size=%d  Execution time=%lf seconds\n\n", K, numElements, numBlocks, blockSize, elapsed_time);






	//256 THREADS NUMBLOCKS  

	blockSize = 256;
	numBlocks = numElements / blockSize + 1; //Since the numElements is not a multiple 256, we add one more block so that there are enough threads for array elements

	// Synchronize to make sure everyone is done in the warmup.
  	hipDeviceSynchronize();

  	// Set up timer
  	clock_gettime(CLOCK_MONOTONIC, &start);

	add<<<numBlocks, blockSize>>> (numElements, x, y);

	// Synchronize to make sure everyone is done.
  	hipDeviceSynchronize();	

	// Compute and report the timing results
  	clock_gettime(CLOCK_MONOTONIC, &end);
	elapsed_time = ((float)end.tv_sec - (float)start.tv_sec) + ((float)end.tv_nsec - (float)start.tv_nsec) / 1000000000.0;

	printf("K=%d Number of elements in array=%d Number of blocks=%d\nBlock size=%d  Execution time=%lf seconds\n\n", K, numElements, numBlocks, blockSize, elapsed_time);




	
	
	hipMemcpy(hy, y, size, hipMemcpyDeviceToHost);

	hipFree(x); 
	hipFree(y); 
	free(hx); 
	free(hy);

	return 0;

} 